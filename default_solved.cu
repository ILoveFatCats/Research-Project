#include "hip/hip_runtime.h"
/*
CUDA - generate array of random numbers and calculate occurence of odd and even numbers - no streams
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define MAX 1000

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

__global__ void generate(unsigned int seed, hiprandState_t* states, int* random) {
  int my_index=blockIdx.x*blockDim.x+threadIdx.x;
  hiprand_init(seed,my_index,0,&states[my_index]);
  random[my_index]=hiprand(&states[my_index]) % MAX;
}

__global__ 
void checkPrime(int *random, int *result) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x;
    int divCount = 0;
    float temp = sqrt((float)random[my_index]);
    for(int i = 2; i < (int)temp;i++)
    {
    	if(random[my_index]%i == 0)
    	{
    		divCount += 1;
    	}
    }
    
   if(divCount == 0)
   {
   atomicAdd(&result[0],1);
   }
   else
   {
   atomicAdd(&result[1],1);
   }
}

__global__ 
void calculateOccurance(int *random, int *hist) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x;
    int index = random[my_index];
   atomicAdd(&hist[index],1);
}


int main(int argc,char **argv) {
    int threadsinblock=1024;
    int blocksingrid=10000;	

    int size = threadsinblock*blocksingrid;

    //memory allocation host
    int *hresults=(int*)malloc(2*sizeof(int));
    int *hrandoms=(int*)malloc(size*sizeof(int));
    int *hhistogram=(int*)malloc(MAX*sizeof(int));

    hiprandState_t* states;

    //memory allocation for generator states
    hipMalloc((void**) &states, size * sizeof(hiprandState_t));

    //memory allocation for randoms
    int *drandom=NULL;
    if (hipSuccess!=hipMalloc((void **)&drandom,size*sizeof(int)))
      errorexit("Error allocating memory on the GPU");

    //memory allocation for results 
    int *dresults=NULL;
    if (hipSuccess!=hipMalloc((void **)&dresults,2*sizeof(int)))
      errorexit("Error allocating memory on the GPU");
      
    int *dhistogram=NULL;
    if (hipSuccess!=hipMalloc((void **)&dhistogram,MAX*sizeof(int)))
      errorexit("Error allocating memory on the GPU");
   
    //results memory initialize with 0
    if (hipSuccess!=hipMemset(dresults,0, 2*sizeof(int)))
      errorexit("Error initializing memory on the GPU");
    
    //hist memory initialize with 0
    if (hipSuccess!=hipMemset(dhistogram,0, MAX*sizeof(int)))
      errorexit("Error initializing memory on the GPU");

    //kernel for number generation
    generate<<<blocksingrid,threadsinblock>>>(time(0),states, drandom);
    if (hipSuccess!=hipGetLastError())
      errorexit("Error during kernel launch");
  
    //copy random numbers to host - i.e. for future to file export
    if (hipSuccess!=hipMemcpy(hrandoms,drandom,size*sizeof(int),hipMemcpyDeviceToHost))
      errorexit("Error copying randoms");
     

    //calculate odd numbers
    checkPrime<<<blocksingrid,threadsinblock>>>(drandom, dresults);
    if (hipSuccess!=hipGetLastError())
      errorexit("Error during kernel launch");

    //calculate even numbers
    calculateOccurance<<<blocksingrid,threadsinblock>>>(drandom, dhistogram);
    
    
        //copy
    if (hipSuccess!=hipMemcpy(hhistogram,dhistogram,MAX*sizeof(int),hipMemcpyDeviceToHost))
      errorexit("Error copying histogram");
      
      
    //copy results to host
    if (hipSuccess!=hipMemcpy(hresults,dresults,2*sizeof(int),hipMemcpyDeviceToHost))
       errorexit("Error copying results");
    
    for(int i = 0; i < MAX; i++)
    {
    printf("number of %d: %d\n", i, hhistogram[i]);
    }
    printf("Found %d prime numbers and %d not prime numbers \n",hresults[0],hresults[1]);

    //free memory
    free(hresults);
    free(hrandoms);
    free(hhistogram);
     if (hipSuccess!=hipFree(states))
      errorexit("Error when deallocating space on the GPU");
    if (hipSuccess!=hipFree(dresults))
      errorexit("Error when deallocating space on the GPU");
    if (hipSuccess!=hipFree(drandom))
      errorexit("Error when deallocating space on the GPU");
    if (hipSuccess!=hipFree(dhistogram))
      errorexit("Error when deallocating space on the GPU");

}

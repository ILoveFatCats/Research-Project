#include "hip/hip_runtime.h"
/*
CUDA - generate array of random numbers and calculate occurence of odd and even numbers - with streams
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define MAX 1000

__host__
void errorexit(const char *s) {
    printf("\n%s",s); 
    exit(EXIT_FAILURE);   
}

__global__ void generate(unsigned int seed, hiprandState_t* states, int* random) {
  int my_index=blockIdx.x*blockDim.x+threadIdx.x;
  hiprand_init(seed,my_index,0,&states[my_index]);
  random[my_index]=hiprand(&states[my_index]) % MAX;
}

__global__ 
void checkPrime(int *random, int *result) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x;
    int divCount = 0;
    float temp = sqrt((float)random[my_index]);
    for(int i = 2; i < (int)temp;i++)
    {
    	if(random[my_index]%i == 0)
    	{
    		divCount += 1;
    	}
    }
    
   if(divCount == 0)
   {
   atomicAdd(&result[0],1);
   }
   else
   {
   atomicAdd(&result[1],1);
   }
}

__global__ 
void calculateOccurance(int *random, int *hist) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x;
    int index = random[my_index];
   atomicAdd(&hist[index],1);
}


int main(int argc,char **argv) {
    int threadsinblock=1024;
    int blocksingrid=10000; 

    int size = threadsinblock*blocksingrid;
    
    //how many streams will be used
    int streamCnt = 3;
    
    //memory allocation host
    int *hresults=NULL;
    hipHostMalloc((void **)&hresults, 2*sizeof(int));
    
    int *hhistogram=NULL;
    hipHostMalloc((void **)&hhistogram, MAX*sizeof(int));
    //int *hhistogram=(int*)malloc(MAX*sizeof(int));


    //int *hresults=(int*)malloc(2*sizeof(int));
    int *hrandoms=(int*)malloc(size*sizeof(int));
    
    //create pointer to streams
    hipStream_t streams[streamCnt];

    hiprandState_t* states;
    //memory allocation for generator states
    hipMalloc((void**) &states, size * sizeof(hiprandState_t));

    //memory allocation for randoms
    int *drandom=NULL;
    if (hipSuccess!=hipMalloc((void **)&drandom,size*sizeof(int)))
      errorexit("Error allocating memory on the GPU");
    
    //memory allocation for results 
    int *dresults=NULL;
    if (hipSuccess!=hipMalloc((void **)&dresults,2*sizeof(int)))
      errorexit("Error allocating memory on the GPU");
      
    // mem alloc for hist
    int *dhistogram=NULL;
    if (hipSuccess!=hipMalloc((void **)&dhistogram,MAX*sizeof(int)))
      errorexit("Error allocating memory on the GPU");

    //results memory initialize with 0
    if (hipSuccess!=hipMemset(dresults,0, 2*sizeof(int)))
      errorexit("Error initializing memory on the GPU");

    //hist memory initialize with 0
    if (hipSuccess!=hipMemset(dhistogram,0, MAX*sizeof(int)))
      errorexit("Error initializing memory on the GPU");
      
    //create streams
    int i;
    for(i=0;i<streamCnt;i++) {
      if (hipSuccess!=hipStreamCreate(&streams[i]))
           errorexit("Error creating stream");
    }

    //generate numbers in stream 0
    generate<<<blocksingrid,threadsinblock, threadsinblock*sizeof(double), streams[0]>>>(time(0),states, drandom);
    
    //oczekiwanie na zakończenie prac przez strumien 0
    hipStreamSynchronize(streams[0]);

    //calculate prime count - stream 1
    checkPrime<<<blocksingrid,threadsinblock,threadsinblock*sizeof(double), streams[1]>>>(drandom, dresults);
    
    //calculate hist - stream 2
    calculateOccurance<<<blocksingrid,threadsinblock, threadsinblock*sizeof(double), streams[2]>>>(drandom, dhistogram);
    
    //asynchronic copy of random numbers
    hipMemcpyAsync(hrandoms,drandom,size*sizeof(int),hipMemcpyDeviceToHost, streams[0]);
    
    //wait for task on streams 1 and 2
    hipStreamSynchronize(streams[1]);
    hipStreamSynchronize(streams[2]);
    
    //asynchronic copy of results from device to host
    if (hipSuccess!=hipMemcpyAsync(hresults,dresults,2*sizeof(int),hipMemcpyDeviceToHost, streams[0]))
       errorexit("Error copying results");
       
    //asynchronic copy of hist from device to host
    if (hipSuccess!=hipMemcpyAsync(hhistogram,dhistogram,MAX*sizeof(int),hipMemcpyDeviceToHost, streams[1]))
       errorexit("Error copying results");
    
    //wait for stream 0 to end its task
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);

    for(int i = 0; i < MAX; i++)
    {
    printf("number of %d: %d\n", i, hhistogram[i]);
    }
    printf("Found %d prime numbers and %d not prime numbers \n",hresults[0],hresults[1]);

    //delete streams
    for(i=0;i<streamCnt;i++) {
      if (hipSuccess!=hipStreamDestroy(*(streams+i)))
         errorexit("Error creating stream");
    }

    
    //free memory
    free(hrandoms);
     if (hipSuccess!=hipHostFree(hresults))
      errorexit("Error when deallocating space on the GPU");
     if (hipSuccess!=hipHostFree(hhistogram))
      errorexit("Error when deallocating space on the GPU");
     if (hipSuccess!=hipFree(states))
      errorexit("Error when deallocating space on the GPU");
    if (hipSuccess!=hipFree(dresults))
      errorexit("Error when deallocating space on the GPU");
    if (hipSuccess!=hipFree(drandom))
      errorexit("Error when deallocating space on the GPU");
    if (hipSuccess!=hipFree(dhistogram))
      errorexit("Error when deallocating space on the GPU");

}
